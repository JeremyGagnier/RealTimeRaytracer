#include "hip/hip_runtime.h"
#include "gpu_hook.h"

void find_dist_to_lens(ray_frame* frame)
{
    float dx = frame->c.camera_width / -2.0f;
    float dy = frame->c.camera_height / -2.0f;
    float relative_x = frame->camera_x.x * dx + frame->camera_y.x * dy;
    float relative_y = frame->camera_x.y * dx + frame->camera_y.y * dy;
    float relative_z = frame->camera_x.z * dx + frame->camera_y.z * dy;

    // Find where the ray hits the lens, then update the ray to equal the normal at that point.
    float o_x = (relative_x + frame->c.direction.x * frame->dist_to_lens) / frame->c.lens_size;
    float o_y = (relative_y + frame->c.direction.y * frame->dist_to_lens) / frame->c.lens_size;
    float o_z = (relative_z + frame->c.direction.z * frame->dist_to_lens) / frame->c.lens_size;
    float d_x = frame->c.direction.x / frame->c.lens_size;
    float d_y = frame->c.direction.y / frame->c.lens_size;
    float d_z = frame->c.direction.z / frame->c.lens_size;

    float dir_dot_dir   = d_x * d_x + d_y * d_y + d_z * d_z;
    float orig_dot_dir  = o_x * d_x + o_y * d_y + o_z * d_z;
    float orig_dot_orig = o_x * o_x + o_y * o_y + o_z * o_z;
	
    float two_a = 2.0f * dir_dot_dir;
    float b = 2.0f * orig_dot_dir;
    float determinant = b * b - 2.0f * two_a * (orig_dot_orig - 1.0f);

    float t = (-b + sqrtf(determinant)) / two_a;	// t has to be the plus det root.

    frame->dist_to_lens = t;
}

__device__ void run_raytracer(int i, vector* colors, ray_frame frame)
{
    ray r;
    float pct_x = ((float)(i % frame.width) + 0.5f) / ((float)frame.width) - 0.5f;
    float pct_y = ((float)(i / frame.width) + 0.5f) / ((float)frame.height) - 0.5f;
    float dx = pct_x * frame.c.camera_width;
    float dy = pct_y * frame.c.camera_height;

    // Do the procedure for each ray at that pixel (14).
    for (int ray_num = 0; ray_num < frame.rays_per_pixel; ++ray_num)
    {
        colors[i].x = 0.0f;
        colors[i].y = 0.0f;
        colors[i].z = 0.0f;

        // Find the origin of the ray (0).
        // TODO: Randomize the camera_x and camera_y variables to cast rays from random points within a pixel
        float relative_x = frame.camera_x.x * dx + frame.camera_y.x * dy;
        float relative_y = frame.camera_x.y * dx + frame.camera_y.y * dy;
        float relative_z = frame.camera_x.z * dx + frame.camera_y.z * dy;
        r.origin.x = frame.c.position.x + relative_x;
        r.origin.y = frame.c.position.y + relative_y;
        r.origin.z = frame.c.position.z + relative_z;

        // Find where the ray hits the lens, then update the ray to equal the normal at that point (12).
        float o_x = (relative_x + frame.c.direction.x * frame.dist_to_lens) / frame.c.lens_size;	// Lens space
        float o_y = (relative_y + frame.c.direction.y * frame.dist_to_lens) / frame.c.lens_size;
        float o_z = (relative_z + frame.c.direction.z * frame.dist_to_lens) / frame.c.lens_size;
        float d_x = frame.c.direction.x / frame.c.lens_size;	// Lens space
        float d_y = frame.c.direction.y / frame.c.lens_size;
        float d_z = frame.c.direction.z / frame.c.lens_size;

        float dir_dot_dir   = d_x * d_x + d_y * d_y + d_z * d_z;
        float orig_dot_dir  = o_x * d_x + o_y * d_y + o_z * d_z;
        float orig_dot_orig = o_x * o_x + o_y * o_y + o_z * o_z;

        float two_a = 2.0f * dir_dot_dir;
        float b = -2.0f * orig_dot_dir;
        float determinant = b * b - 2.0f * two_a * (orig_dot_orig - 1.0f);

        float t = (b + __fsqrt_rn(determinant)) / two_a;	// t has to be the plus det root.

        // Update the origin to the surface of the sphere (48 + 1sqrt).
        r.origin.x += frame.c.direction.x * t;
        r.origin.y += frame.c.direction.y * t;
        r.origin.z += frame.c.direction.z * t;

        // Update the direction to be the center of the lens to the surface (51 + 1sqrt)
        r.direction.x = o_x + d_x * t;
        r.direction.y = o_y + d_y * t;
        r.direction.z = o_z + d_z * t;

        float shortestTime = INT_MAX;   // I don't see a float max...
        triangle* hitTri = NULL;
        triangle* trisEnd = frame.tris + frame.numTris;
        for (triangle* triIterator = frame.tris; triIterator < trisEnd; ++triIterator)
        {
            if (hit_tri(r, *triIterator, &shortestTime))
            {
                hitTri = triIterator;
            }
        }
        
        if (hitTri != NULL)
        {
            r.origin.x += r.direction.x * shortestTime;
            r.origin.y += r.direction.y * shortestTime;
            r.origin.z += r.direction.z * shortestTime;
            /*for ()
            {

            }*/
            // TEST
            colors[i].x = hitTri->m->mat.color.x;
            colors[i].y = hitTri->m->mat.color.y;
            colors[i].z = hitTri->m->mat.color.z;
        }
    }
}

// Worst case of 63 operations. Good chance of 18 operations. Very low chance of 8 operations.
__device__ bool hit_tri(ray r, triangle t, float* shortest_time)
{
    float denom = t.normal.x * r.direction.x + t.normal.y * r.direction.y + t.normal.z * r.direction.z;
    if (denom < SMALL_FLOAT & denom > -SMALL_FLOAT)
    {
        // Hit edge on, very low chance (8).
        return false;
    }

    float time = (t.normal.x * (t.origin.x - r.origin.x) + t.normal.y * (t.origin.y - r.origin.y) + t.normal.z * (t.origin.z - r.origin.z)) / denom;
    if (*shortest_time <= time)
    {
        // Can't possibly hit earlier, good chance (~50%) (18).
        return false;
    }

    float hit_point_x = r.origin.x + time * r.direction.x;
    float hit_point_y = r.origin.y + time * r.direction.y;
    float hit_point_z = r.origin.z + time * r.direction.z;
    float wx = hit_point_x - t.origin.x;
    float wy = hit_point_y - t.origin.y;
    float wz = hit_point_z - t.origin.z;
    
    float s_dot_t = t.s.x * t.t.x + t.s.y * t.t.y + t.s.z * t.t.z;
    float s_dot_s = t.s.x * t.s.x + t.s.y * t.s.y + t.s.z * t.s.z;
    float t_dot_t = t.t.x * t.t.x + t.t.y * t.t.y + t.t.z * t.t.z;
    float w_dot_s = wx    * t.s.x + wy    * t.s.y + wz    * t.s.z;
    float w_dot_t = wx    * t.t.x + wy    * t.t.y + wz    * t.t.z;

    denom = s_dot_t * s_dot_t - s_dot_s * t_dot_t;
    float d1 = (s_dot_t * w_dot_t - t_dot_t * w_dot_s) / denom;
    float d2 = (s_dot_t * w_dot_s - s_dot_s * w_dot_t) / denom;

    // Check that the ray passes through the triangle, low chance (53).
    if ((0.0f <= d1) & (d1 <= 1.0f) & (0.0f <= d2) & (d2 <= 1.0f) & ((d1 + d2) <= 1.0f))
    {
        *shortest_time = time;
        return true;
    }
    return false;
}
