#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void kernelWrapper(vector* colors, ray_frame* frame)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    colors[i].x = 0.0f;
	colors[i].y = 0.0f;
	colors[i].z = 0.0f;
    run_raytracer(i, colors, *frame);
}

void checkErrors() {
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
}

int kernel_main()
{
    // Set up inputs --------------------------------------------------------------------------------------------------
    vector cameraPosition = {0.0f, 0.0f, 0.0f};
    vector cameraDirection = {0.0f, 0.0f, 1.0f};
    float cameraWidth = 1.0f;
    float cameraHeight = 0.5625f;
    float lensSize = 3.0f;

    // TODO: Load meshes from files
    mesh red_mesh  = {0, {{1.0f, 0.0f, 0.0f}, 0.0f, {0.0f, 0.5f, 0.5f}}};
    mesh blue_mesh = {0, {{0.0f, 1.0f, 0.0f}, 0.0f, {0.5f, 0.0f, 0.5f}}};
    // TODO: Compute tris from meshes
    const int numTris = 2;
    triangle tris[numTris] =
    {
        {{-1.0f, -1.0f, 10.0f}, {0.0f, 10.0f, 0.0f}, {10.0f, 0.0f, 0.0f}, {0, 0, -1}, 0},
        {{1.0f, 1.0f, 10.0f}, {0.0f, -10.0f, 0.0f}, {-10.0f, 0.0f, 0.0f}, {0, 0, -1}, 0}
    };

    int width = 16;		// 1.0000
    int height = 9;		// 0.5625
    int rays_per_pixel = 1;
    int depth = 1;
	
    // Build the core datastructures ----------------------------------------------------------------------------------
    camera camera = 
    {
        cameraPosition,
        cameraDirection,
        cameraWidth,
        cameraHeight,
        lensSize
    };
	vector up = {0.0f, 1.0f, 0.0f}; // TODO: Make sure that the camera isn't parallel to up
	vector camera_x = {up.y * cameraDirection.z - up.z * cameraDirection.y,
					   up.z * cameraDirection.x - up.x * cameraDirection.z,
					   up.x * cameraDirection.y - up.y * cameraDirection.x};
	vector camera_y = {cameraDirection.y * camera_x.z - cameraDirection.z * camera_x.y,
					   cameraDirection.z * camera_x.x - cameraDirection.x * camera_x.z,
					   cameraDirection.x * camera_x.y - cameraDirection.y * camera_x.x};
    ray_frame frame = 
	{
		camera,
        numTris,
		0,      // This reference will be set in the GPU
		width,
        height,
		rays_per_pixel,
        depth,
		up,
        camera_x,
        camera_y,
		0.0f
	};
	find_dist_to_lens(&frame);

    // Allocate memory in the GPU -------------------------------------------------------------------------------------
    mesh* red_mesh_ptr;
    mesh* blue_mesh_ptr;
    triangle* tris_ptr;
    vector* colors_ptr;
    ray_frame* frame_ptr;

    hipSetDevice(0);
    hipMalloc((void**)&red_mesh_ptr, sizeof(mesh));
    hipMalloc((void**)&blue_mesh_ptr, sizeof(mesh));
    hipMalloc((void**)&tris_ptr, sizeof(triangle) * numTris);
    hipMalloc((void**)&colors_ptr, sizeof(vector) * width * height);
    hipMalloc((void**)&frame_ptr, sizeof(ray_frame));

    tris[0].m = red_mesh_ptr;
    tris[1].m = blue_mesh_ptr;
    frame.tris = tris_ptr;

    hipMemcpy(red_mesh_ptr, &red_mesh, sizeof(mesh), hipMemcpyHostToDevice);
    hipMemcpy(blue_mesh_ptr, &blue_mesh, sizeof(mesh), hipMemcpyHostToDevice);
    hipMemcpy(tris_ptr, tris, sizeof(triangle) * numTris, hipMemcpyHostToDevice);
    hipMemcpy(frame_ptr, &frame, sizeof(ray_frame), hipMemcpyHostToDevice);

    // Run the kernal -------------------------------------------------------------------------------------------------
    int threads = height;
    int blocks = width;
    if (threads > 1024)
    {
        threads /= 2;
        blocks *= 2;
    }
    kernelWrapper<<<blocks, threads>>>(colors_ptr, frame_ptr);
    checkErrors();

    vector* colors = (vector*)malloc(sizeof(vector) * width * height);
    hipMemcpy(colors, colors_ptr, sizeof(vector) * width * height, hipMemcpyDeviceToHost);

    // hipDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual
    // Profiler to show complete traces.
    hipDeviceReset();

    
	for (int i = 0; i < width * height; ++i)
	{
        if (true)//(colors[i].x != 0.0f || colors[i].y != 0.0f || colors[i].z != 0.0f)
        {
		    fprintf(stdout, "(%f, %f, %f)\n", colors[i].x, colors[i].y, colors[i].z);
        }
	}

    hipFree(frame_ptr);
    hipFree(colors_ptr);
    hipFree(tris_ptr);
    hipFree(blue_mesh_ptr);
    hipFree(red_mesh_ptr);
    
	system("pause");
    return 0;
}
